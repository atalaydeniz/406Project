#include <iostream>
#include <hip/hip_runtime.h>
#include <random>
#include <omp.h>

#define TILE_SIZE 16

__global__ void matrixMultiplication(const float* matrix1, const float* matrix2, float* result,
                                     int rows1, int cols1, int cols2) {
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    int row = threadIdx.y;
    int col = threadIdx.x;

    int globalRow = blockRow * blockDim.y + row;
    int globalCol = blockCol * blockDim.x + col;

    __shared__ float sharedMatrix1[TILE_SIZE][TILE_SIZE];
    __shared__ float sharedMatrix2[TILE_SIZE][TILE_SIZE];

    float sum = 0.0f;
    int numTiles = (cols1 + TILE_SIZE - 1) / TILE_SIZE;

    for (int t = 0; t < numTiles; t++) {
        if (globalRow < rows1 && t * TILE_SIZE + col < cols1) {
            sharedMatrix1[row][col] = matrix1[globalRow * cols1 + t * TILE_SIZE + col];
        } else {
            sharedMatrix1[row][col] = 0.0f;
        }

        if (t * TILE_SIZE + row < cols1 && globalCol < cols2) {
            sharedMatrix2[row][col] = matrix2[(t * TILE_SIZE + row) * cols2 + globalCol];
        } else {
            sharedMatrix2[row][col] = 0.0f;
        }

        __syncthreads();

        for (int i = 0; i < TILE_SIZE; i++) {
            sum += sharedMatrix1[row][i] * sharedMatrix2[i][col];
        }

        __syncthreads();
    }

    if (globalRow < rows1 && globalCol < cols2) {
        result[globalRow * cols2 + globalCol] = sum;
    }
}

int main() {
    int rows1 = 16384;
    int cols1 = 16384;
    int cols2 = 16384;

    // Matrix sizes
    int matrix1Size = rows1 * cols1 * sizeof(float);
    int matrix2Size = cols1 * cols2 * sizeof(float);
    int resultSize = rows1 * cols2 * sizeof(float);

    // Host matrices
    float* h_matrix1 = (float*)malloc(matrix1Size);
    float* h_matrix2 = (float*)malloc(matrix2Size);
    float* h_result = (float*)malloc(resultSize);

    double lower_bound = 0;
    double upper_bound = 100;
    std::uniform_real_distribution<double> unif(lower_bound, upper_bound);
    std::default_random_engine re;

    for (int i = 0; i < 16384*16384; i++) {
            double r = unif(re);
            h_matrix1[i] = r;
            r = unif(re);
            h_matrix2[i] = r;
            h_result[i] = 0;
    }
    
    // Device matrices
    float* d_matrix1;
    float* d_matrix2;
    float* d_result;

    // Allocate device memory
    hipMalloc((void**)&d_matrix1, matrix1Size);
    hipMalloc((void**)&d_matrix2, matrix2Size);
    hipMalloc((void**)&d_result, resultSize);

    // Transfer data from host to device
    hipMemcpy(d_matrix1, h_matrix1, matrix1Size, hipMemcpyHostToDevice);
    hipMemcpy(d_matrix2, h_matrix2, matrix2Size, hipMemcpyHostToDevice);

    // Define block and grid dimensions
    dim3 blockSize(4, 4);
    dim3 gridSize((cols2 + blockSize.x - 1) / blockSize.x, (rows1 + blockSize.y - 1) / blockSize.y);

    // Launch the kernel
    double s = omp_get_wtime();
    matrixMultiplication<<<gridSize, blockSize>>>(d_matrix1, d_matrix2, d_result, rows1, cols1, cols2);
    double e = omp_get_wtime();
    std::cout << "Running Time: " << e - s << std::endl;
    // Transfer result from device to host
    hipMemcpy(h_result, d_result, resultSize, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_matrix1);
    hipFree(d_matrix2);
    hipFree(d_result);

    // Free host memory
    free(h_matrix1);
    free(h_matrix2);
    free(h_result);

    return 0;
}
