#include <iostream>
#include <hip/hip_runtime.h>
#include <random>
#include <omp.h>

__global__ void matrixMultiplication(const float* matrix1, const float* matrix2, float* result,
                                     int rows1, int cols1, int cols2) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows1 && col < cols2) {
        float sum = 0.0f;
        for (int i = 0; i < cols1; i++) {
            sum += matrix1[row * cols1 + i] * matrix2[i * cols2 + col];
        }
        result[row * cols2 + col] = sum;
    }
}

int main() {
    int rows1 = 16384;
    int cols1 = 16384;
    int cols2 = 16384;

    // Matrix sizes
    int matrix1Size = rows1 * cols1 * sizeof(float);
    int matrix2Size = cols1 * cols2 * sizeof(float);
    int resultSize = rows1 * cols2 * sizeof(float);

    // Host matrices
    float* h_matrix1 = (float*)malloc(matrix1Size);
    float* h_matrix2 = (float*)malloc(matrix2Size);
    float* h_result = (float*)malloc(resultSize);

    double lower_bound = 0;
    double upper_bound = 100;
    std::uniform_real_distribution<double> unif(lower_bound, upper_bound);
    std::default_random_engine re;

    for (int i = 0; i < 16384*16384; i++) {
            double r = unif(re);
            h_matrix1[i] = r;
            r = unif(re);
            h_matrix2[i] = r;
            h_result[i] = 0;
    }
    
    // Device matrices
    float* d_matrix1;
    float* d_matrix2;
    float* d_result;

    // Allocate device memory
    hipMalloc((void**)&d_matrix1, matrix1Size);
    hipMalloc((void**)&d_matrix2, matrix2Size);
    hipMalloc((void**)&d_result, resultSize);

    // Transfer data from host to device
    hipMemcpy(d_matrix1, h_matrix1, matrix1Size, hipMemcpyHostToDevice);
    hipMemcpy(d_matrix2, h_matrix2, matrix2Size, hipMemcpyHostToDevice);

    // Define block and grid dimensions
    dim3 blockSize(4, 4);
    dim3 gridSize((cols2 + blockSize.x - 1) / blockSize.x, (rows1 + blockSize.y - 1) / blockSize.y);

    // Launch the kernel
    double s = omp_get_wtime();
    matrixMultiplication<<<gridSize, blockSize>>>(d_matrix1, d_matrix2, d_result, rows1, cols1, cols2);
    double e = omp_get_wtime();
    std::cout << "Running Time: " << e - s << std::endl;
    // Transfer result from device to host
    hipMemcpy(h_result, d_result, resultSize, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_matrix1);
    hipFree(d_matrix2);
    hipFree(d_result);

    // Free host memory
    free(h_matrix1);
    free(h_matrix2);
    free(h_result);

    return 0;
}
