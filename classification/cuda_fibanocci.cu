/*

Classified as "Algorithm/Data Stucture" --> "Inefficient Algorithm"

*/


#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

__global__ void fibonacciCUDA(int* fibSeq, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n) {
        if (tid <= 1) {
            fibSeq[tid] = tid;
        } else {
            fibSeq[tid] = fibSeq[tid - 1] + fibSeq[tid - 2];
        }
    }
}

int main() {
    int n = 100;  // Number of Fibonacci numbers to calculate

    // Allocate memory on the host for the Fibonacci sequence
    int* hostFibSeq = new int[n];

    // Allocate memory on the device (GPU) for the Fibonacci sequence
    int* deviceFibSeq;
    hipMalloc((void**)&deviceFibSeq, n * sizeof(int));

    // Set up the kernel launch configuration
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    // Launch the CUDA kernel
    fibonacciCUDA<<<gridSize, blockSize>>>(deviceFibSeq, n);

    // Copy the result from the device to the host
    hipMemcpy(hostFibSeq, deviceFibSeq, n * sizeof(int), hipMemcpyDeviceToHost);

    // Print the Fibonacci sequence
    std::cout << "Fibonacci Sequence: ";
    for (int i = 0; i < n; i++) {
        std::cout << hostFibSeq[i] << " ";
    }
    std::cout << std::endl;

    // Free memory
    delete[] hostFibSeq;
    hipFree(deviceFibSeq);

    return 0;
}
