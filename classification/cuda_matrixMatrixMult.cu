/*

Classified as "Inefficient Parallelization" --> "Inefficient Thread Mapping"
              "Micro-Architectural" --> "Data Locality"

*/


#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

__global__ void matrixMatrixMultiplication_cuda(const int* matrix1, const int* matrix2, int* result, int matrix1Rows, int matrix1Columns, int matrix2Columns) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < matrix1Rows && col < matrix2Columns) {
        int value = 0;
        for (int i = 0; i < matrix1Columns; i++) {
            value += matrix1[row * matrix1Columns + i] * matrix2[i * matrix2Columns + col];
        }

        result[row * matrix2Columns + col] = value;
    }
}

std::vector<std::vector<int>> matrixMatrixMultiplication_cuda(const std::vector<std::vector<int>>& matrix1, const std::vector<std::vector<int>>& matrix2) {
    int matrix1Rows = matrix1.size();
    int matrix1Columns = matrix1[0].size();
    int matrix2Rows = matrix2.size();
    int matrix2Columns = matrix2[0].size();

    if (matrix1Columns != matrix2Rows) {
        std::cout << "Error: Matrix1 columns and Matrix2 rows must match." << std::endl;
        return std::vector<std::vector<int>>();
    }

    std::vector<int> matrix1_flat(matrix1Rows * matrix1Columns);
    std::vector<int> matrix2_flat(matrix2Rows * matrix2Columns);
    std::vector<int> result_flat(matrix1Rows * matrix2Columns);

    for (int i = 0; i < matrix1Rows; i++) {
        for (int j = 0; j < matrix1Columns; j++) {
            matrix1_flat[i * matrix1Columns + j] = matrix1[i][j];
        }
    }

    for (int i = 0; i < matrix2Rows; i++) {
        for (int j = 0; j < matrix2Columns; j++) {
            matrix2_flat[i * matrix2Columns + j] = matrix2[i][j];
        }
    }

    int* device_matrix1;
    int* device_matrix2;
    int* device_result;

    hipMalloc((void**)&device_matrix1, matrix1Rows * matrix1Columns * sizeof(int));
    hipMalloc((void**)&device_matrix2, matrix2Rows * matrix2Columns * sizeof(int));
    hipMalloc((void**)&device_result, matrix1Rows * matrix2Columns * sizeof(int));

    hipMemcpy(device_matrix1, matrix1_flat.data(), matrix1Rows * matrix1Columns * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_matrix2, matrix2_flat.data(), matrix2Rows * matrix2Columns * sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((matrix2Columns + threadsPerBlock.x - 1) / threadsPerBlock.x, (matrix1Rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrixMatrixMultiplication_cuda<<<numBlocks, threadsPerBlock>>>(device_matrix1, device_matrix2, device_result, matrix1Rows, matrix1Columns, matrix2Columns);

    hipMemcpy(result_flat.data(), device_result, matrix1Rows * matrix2Columns * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_matrix1);
    hipFree(device_matrix2);
    hipFree(device_result);

    std::vector<std::vector<int>> result(matrix1Rows, std::vector<int>(matrix2Columns));

    for (int i = 0; i < matrix1Rows; i++) {
        for (int j = 0; j < matrix2Columns; j++) {
            result[i][j] = result_flat[i * matrix2Columns + j];
        }
    }

    return result;
}

int main() {
    std::vector<std::vector<int>> matrix1 = {{1, 2, 3}, {4, 5, 6}};
    std::vector<std::vector<int>> matrix2 = {{7, 8}, {9, 10}, {11, 12}};

    std::vector<std::vector<int>> result = matrixMatrixMultiplication_cuda(matrix1, matrix2);

    std::cout << "Result: " << std::endl;
    for (const auto& row : result) {
        for (int val : row) {
            std::cout << val << " ";
        }
        std::cout << std::endl;
    }

    return 0;
}
