/*

Classified as "Memory Management" --> "Memory Leak"
              "Micro-Architectural" --> "Data Locality"
              "Inefficient Parallelization" --> "Underparallelization"

*/


#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

__global__ void matrixAdditionCUDA(const int* matrix1, const int* matrix2, int* result, int rows, int columns) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < columns) {
        result[row * columns + col] = matrix1[row * columns + col] + matrix2[row * columns + col];
    }
}

std::vector<std::vector<int>> matrixAdditionCUDA(const std::vector<std::vector<int>>& matrix1, const std::vector<std::vector<int>>& matrix2) {
    int rows = matrix1.size();
    int columns = matrix1[0].size();

    std::vector<int> matrix1_flat(rows * columns);
    std::vector<int> matrix2_flat(rows * columns);
    std::vector<int> result_flat(rows * columns);

    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < columns; j++) {
            matrix1_flat[i * columns + j] = matrix1[i][j];
            matrix2_flat[i * columns + j] = matrix2[i][j];
        }
    }

    int* device_matrix1;
    int* device_matrix2;
    int* device_result;

    hipMalloc((void**)&device_matrix1, rows * columns * sizeof(int));
    hipMalloc((void**)&device_matrix2, rows * columns * sizeof(int));
    hipMalloc((void**)&device_result, rows * columns * sizeof(int));

    hipMemcpy(device_matrix1, matrix1_flat.data(), rows * columns * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_matrix2, matrix2_flat.data(), rows * columns * sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((columns + threadsPerBlock.x - 1) / threadsPerBlock.x, (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrixAdditionCUDA<<<numBlocks, threadsPerBlock>>>(device_matrix1, device_matrix2, device_result, rows, columns);

    hipMemcpy(result_flat.data(), device_result, rows * columns * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_matrix1);
    hipFree(device_matrix2);
    hipFree(device_result);

    std::vector<std::vector<int>> result(rows, std::vector<int>(columns));

    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < columns; j++) {
            result[i][j] = result_flat[i * columns + j];
        }
    }

    return result;
}

int main() {
    std::vector<std::vector<int>> matrix1 = {{1, 2, 3}, {4, 5, 6}};
    std::vector<std::vector<int>> matrix2 = {{7, 8, 9}, {10, 11, 12}};

    std::vector<std::vector<int>> result = matrixAdditionCUDA(matrix1, matrix2);

    std::cout << "Result: " << std::endl;
    for (const auto& row : result) {
        for (int val : row) {
            std::cout << val << " ";
        }
        std::cout << std::endl;
    }

    return 0;
}
